#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/opencv.hpp>

__device__ uint8_t median_pixel(uint8_t *picture, size_t n_row_window, size_t n_col_window, size_t n_col_picture) {
  uint32_t counting[256];
  memset(counting, 0 ,sizeof(uint32_t)*256);
  uint8_t pixel_picture=0;
  for(int i=0;i<n_row_window;++i)
	for (int j=0;j<n_col_window;++j){
	  pixel_picture = picture[(i*n_col_picture+j)*3];
	  counting[pixel_picture]+=1;
	}
  uint32_t median_border = (n_row_window*n_col_window-1)/2;
  uint32_t tmp=0;
  uint8_t pixel_median=-1;
  while(tmp<median_border){
	pixel_median+=1;
	tmp+=counting[pixel_median];
  }
  return pixel_median;
}

// kernel
__global__ void filter_median_apply_hsv(uint8_t* picture, uint8_t* picture_blur) {
  size_t median_col = 5;
  size_t median_row = 5;

  size_t i = blockIdx.x;
  size_t j = threadIdx.x;
  size_t window_upper = i-(median_row-1)/2;
  size_t window_lower = i+(median_row-1)/2;
  size_t window_left = j-(median_col-1)/2;
  size_t window_right = j+(median_col-1)/2;
  size_t pos_pixel = i*blockDim.x+j;

  if ((window_upper<0)
	  |((window_lower+1)>gridDim.x)
	  |(window_left<0)
	  |((window_right+1)>blockDim.x)){
	picture_blur[pos_pixel*3] = picture[pos_pixel*3];
	picture_blur[pos_pixel*3+1] = picture[pos_pixel*3+1];
	picture_blur[pos_pixel*3+2] = picture[pos_pixel*3+2];
  }
  picture_blur[pos_pixel*3] = median_pixel(&picture[(window_upper*blockDim.x+window_left)*3], median_row, median_col, blockDim.x);
  picture_blur[pos_pixel*3+1] = median_pixel(&picture[(window_upper*blockDim.x+window_left)*3+1], median_row, median_col, blockDim.x);
  picture_blur[pos_pixel*3+2] = median_pixel(&picture[(window_upper*blockDim.x+window_left)*3+2], median_row, median_col, blockDim.x);
  __syncthreads();
}

__host__ void filter_median_hsv(const uint8_t *h_input, uint8_t *h_output, size_t n_row, size_t n_col) {
  size_t size = sizeof(uint8_t) * n_row * n_col * 3;
  uint8_t *d_input;
  hipMalloc(&d_input, size);
  hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice);

  uint8_t *d_output;
  hipMalloc(&d_output, size);
  hipMemset(d_output, 0, size);

  filter_median_apply_hsv<<<
	  dim3(n_row, 1, 1),
	  dim3(n_col, 1, 1)>>>(d_input, d_output);

  hipMemcpy(h_output, d_output, size, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  hipFree(d_input);
  hipFree(d_output);
}