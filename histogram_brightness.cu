#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/opencv.hpp>

// kernel
__global__ void calculate_histogram_brightness_hsv_kernel(const uint8_t* picture, uint32_t* histogram_brightness) {
  size_t i = blockIdx.x;
  size_t j = threadIdx.x;

  uint8_t brightness = picture[(i*blockDim.x+j)*3+2];
  histogram_brightness[brightness] += 1;
  __syncthreads();
}



__host__ void calculate_histogram_brightness_hsv(const uint8_t* h_input, uint32_t* h_histogram, size_t n_row, size_t n_col) {
  size_t size_input = sizeof(uint8_t) * n_row * n_col * 3;
  size_t size_output = sizeof(uint32_t)*256;
  uint8_t* d_input;
  hipMalloc(&d_input, size_input);
  hipMemcpy(d_input, h_input, size_input, hipMemcpyHostToDevice);

  uint32_t* d_output;
  hipMalloc(&d_output, size_output);
  hipMemset(d_output, 0, size_output);

  calculate_histogram_brightness_hsv_kernel<<<
  dim3(n_row,1,1),
  dim3(n_col, 1, 1)
  >>>(d_input, d_output);

  hipMemcpy(h_histogram, d_output, size_output, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  hipFree(d_input);
  hipFree(d_output);
}