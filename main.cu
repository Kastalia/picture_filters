#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <iostream>


void filter_linear_bgr(const uint8_t* h_input, uint8_t* h_output, size_t n_row, size_t n_col);
void filter_median_hsv(const uint8_t* h_input, uint8_t* h_output, size_t n_row, size_t n_col);
void calculate_histogram_brightness_hsv(const uint8_t* h_input, uint32_t* h_histogram, size_t n_row, size_t n_col);

int main() {
  cv::Mat pic = cv::imread("../partyhard.jpg",cv::IMREAD_COLOR);
  cv::imshow("partyhard_original", pic);
  cv::Mat pic_hsv;
  cvtColor(pic, pic_hsv,CV_BGR2HSV);
  cv::Mat pic_blur, pic_blur_hsv;
  clock_t start_s, stop_s;


  // linear filter
  pic_blur = cv::Mat::zeros(cv::Size(pic.cols, pic.rows), CV_8UC3);

  start_s = clock();
  filter_linear_bgr(pic.data, pic_blur.data, pic.rows, pic.cols);
  stop_s = clock();
  std::cout << "Time for the GPU blur linear filter: " << (stop_s - start_s) / double(CLOCKS_PER_SEC) * 1000 <<  " ms" << std::endl;

  cv::imshow("partyhard_filter_linear", pic_blur);


  //median filter
  pic_blur = cv::Mat::zeros(cv::Size(pic_hsv.cols, pic_hsv.rows), CV_8UC3);
  pic_blur_hsv = cv::Mat::zeros(cv::Size(pic_hsv.cols, pic_hsv.rows), CV_8UC3);

  start_s = clock();
  filter_median_hsv(pic_hsv.data, pic_blur_hsv.data, pic_hsv.rows, pic_hsv.cols);
  stop_s = clock();
  std::cout << "Time for the GPU blur median filter: " << (stop_s - start_s) / double(CLOCKS_PER_SEC) * 1000 <<  " ms" << std::endl;

  cvtColor(pic_blur_hsv, pic_blur,CV_HSV2BGR);
  cv::imshow("partyhard_filter_median", pic_blur);


  // histogram brightness. Построил по значению brightness в HSV представлении изображения.
  uint32_t hist_brightness[256];
  memset(hist_brightness, 0 , sizeof(uint32_t)*256);

  start_s = clock();
  calculate_histogram_brightness_hsv(pic_hsv.data, hist_brightness, pic_hsv.rows, pic_hsv.cols);
  stop_s = clock();
  std::cout << "Time for the GPU calculate histogram brightness: " << (stop_s - start_s) / double(CLOCKS_PER_SEC) * 1000 <<  " ms" << std::endl;

  int32_t sum_pixels=0;
  for(auto& elem:hist_brightness){
	std::cout<<elem<<" "<<std::endl;
	sum_pixels+=elem;
  }
  std::cout<<"Сумма="<<sum_pixels<<std::endl;
  cv::waitKey();
}