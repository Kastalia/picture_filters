#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <iostream>


void filter_linear(const uint8_t* h_input, uint8_t* h_output, size_t n_row, size_t n_col);

int main() {
  std::cout<<"its cpu time\n";

  cv::Mat pic = cv::imread("../partyhard.jpg",cv::IMREAD_COLOR);
  cv::Mat pic_blur = cv::Mat::zeros(cv::Size(pic.cols, pic.rows), CV_8UC3);

  clock_t start_s = clock();
  filter_linear(pic.data, pic_blur.data, pic.rows, pic.cols);
  clock_t stop_s = clock();
  std::cout << "Time for the GPU blur linear filter: " << (stop_s - start_s) / double(CLOCKS_PER_SEC) * 1000 <<  " ms" << std::endl;

  cv::imshow("partyhard_original", pic);
  cv::imshow("partyhard_filter_linear", pic_blur);
  cv::waitKey();

}