#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <iostream>


void filter_linear_bgr(const uint8_t* h_input, uint8_t* h_output, size_t n_row, size_t n_col);
void filter_median_hsv(const uint8_t* h_input, uint8_t* h_output, size_t n_row, size_t n_col);

int main() {
  std::cout<<"its cpu time\n";

  cv::Mat pic = cv::imread("../partyhard.jpg",cv::IMREAD_COLOR);

  // linear filter
  cv::Mat pic_blur = cv::Mat::zeros(cv::Size(pic.cols, pic.rows), CV_8UC3);
  /*
  clock_t start_s = clock();
  filter_linear_bgr(pic.data, pic_blur.data, pic.rows, pic.cols);
  clock_t stop_s = clock();
  std::cout << "Time for the GPU blur linear filter: " << (stop_s - start_s) / double(CLOCKS_PER_SEC) * 1000 <<  " ms" << std::endl;

  cv::imshow("partyhard_original", pic);
  cv::imshow("partyhard_filter_linear", pic_blur);
  cv::waitKey();
   */


  cv::Mat pic_hsv;
  cvtColor(pic, pic_hsv,CV_BGR2HSV);
  cv::Mat pic_blur_hsv = cv::Mat::zeros(cv::Size(pic_hsv.cols, pic_hsv.rows), CV_8UC3);
  pic_blur = cv::Mat::zeros(cv::Size(pic_hsv.cols, pic_hsv.rows), CV_8UC3);

  clock_t start_s = clock();
  filter_median_hsv(pic_hsv.data, pic_blur_hsv.data, pic_hsv.rows, pic_hsv.cols);
  clock_t stop_s = clock();
  std::cout << "Time for the GPU blur median filter: " << (stop_s - start_s) / double(CLOCKS_PER_SEC) * 1000 <<  " ms" << std::endl;

  cvtColor(pic_blur_hsv, pic_blur,CV_HSV2BGR);
  cv::imshow("partyhard_original", pic);
  cv::imshow("partyhard_filter_median", pic_blur);

  cv::waitKey();


}