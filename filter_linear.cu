#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/opencv.hpp>

// kernel
__global__ void filter_linear_apply(uint8_t* picture, uint8_t* picture_blur) {
  size_t sum_filter = 0;
  uint8_t filter[25];
  for (auto &f : filter) {
	f = 1;
	sum_filter += f;
  }
  size_t i = blockIdx.x;
  size_t j = threadIdx.x;
  uint8_t partpic_blue;
  uint8_t partpic_green;
  uint8_t partpic_red;
  size_t pos_pixel = 0;

  //picture_blur[3 * i * blockDim.x + 3 * j] = 255;
  //picture_blur[3 * i * blockDim.x + 3 * j + 1] = 255;
  //picture_blur[3 * i * blockDim.x + 3 * j + 2] = 0;
  //printf("%d ",picture[3 * i * blockDim.x + 3 * j + 2]);

  //printf("gDim.x_rows_=%d  bDim.x_columns_=%d bIdx.x_i_=%d  tIdx.x_j_=%d pic_blue=%d filter_blue=%d\n", gridDim.x, blockDim.x, blockIdx.x, threadIdx.x, picture[3 * i * blockDim.x + 3 * j], picture_blur[3 * i * blockDim.x + 3 * j]);

  for(int k=-2;k<=2; k++)
  {
	for(int l=-2; l<=2; l++)
	{
	  if(((i+k+1)>gridDim.x)|((i+k)<0)|((j+l+1)>blockDim.x)|((j+l)<0)) {
		//вышли за рамки, будем усредняться по преобразуемому пикселю
		pos_pixel = i * blockDim.x + j;
		partpic_blue = picture[pos_pixel*3];
		partpic_green = picture[pos_pixel*3+1];
		partpic_red = picture[pos_pixel*3+2];
	  }
	  else {
		pos_pixel = (i + k) * blockDim.x + j + l;
		partpic_blue = picture[pos_pixel*3];
		partpic_green= picture[pos_pixel*3+1];
		partpic_red= picture[pos_pixel*3+2];
	  }
	  pos_pixel = i*blockDim.x+j;
	  picture_blur[pos_pixel*3]+=1.0/sum_filter*filter[(k+2)*5+l+2]*partpic_blue;
	  picture_blur[pos_pixel*3+1]+=1.0/sum_filter*filter[(k+2)*5+l+2]*partpic_green;
	  picture_blur[pos_pixel*3+2]+=1.0/sum_filter*filter[(k+2)*5+l+2]*partpic_red;
	};
  };
  /*
   * GUIDE:
  for(int k=-2;k<=2; k++)
  {
	for(int l=-2; l<=2; l++)
	{
	  u[i][j]+=1/s*a[k][l]*u[i+k][j+l];
	}
  }
  hipDeviceSynchronize();
  */
  __syncthreads();

}



__host__ void filter_linear(const uint8_t* h_input, uint8_t* h_output, size_t n_row, size_t n_col) {
  size_t size = sizeof(uint8_t) * n_row * n_col * 3;
  uint8_t* d_input;
  hipMalloc(&d_input, size);
  hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice);

  uint8_t* d_output;
  hipMalloc(&d_output, size);
  hipMemset(d_output, 0, size);

  filter_linear_apply<<<
  dim3(n_row,1,1),
  dim3(n_col, 1, 1)
  >>>(d_input, d_output);

  hipMemcpy(h_output, d_output, size, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  hipFree(d_input);
  hipFree(d_output);
}